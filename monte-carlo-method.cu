#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>


// Perform following sequence of tasks in order:
// 1. Allocate GPU memory to store the random number and simulation path results.
// 2. Call cuRand library to generate random numbers.
// 3. Launch the barrier option kernel to do parallel simulations.
// 4. Launch the sum kernel to aggregate the terminal underlying asset prices.
// 5. Deallocate the memory

// Asian Barrier Option Conditions:
  // Stock Price (K): 110
  // Barrier Price (B): 100
  // Spot Price (SO): 120
  // Sigma: 0.35
  // Percent Drift (mu): 0.1
  // Interest Rate (r): 0.05
  // Maturity: 1 year

// Expected Result(s) // V100 GPU:
  // Option Price: $18.70
  // Time: 26.6ms
  // Paths: 8.192M
  // Steps: 365

#define CHECKCURAND(expression)                         \
  {                                                     \
    hiprandStatus_t status = (expression);                         \
    if (status != HIPRAND_STATUS_SUCCESS) {                        \
      std::cerr << "Curand Error on line " << __LINE__<< std::endl;     \
      std::exit(EXIT_FAILURE);                                          \
    }                                                                   \
  }
 
// atomicAdd is introduced for compute capability >=6.0
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
{
      printf("device arch <=600\n");
        unsigned long long int* address_as_ull = (unsigned long long int*)address;
          unsigned long long int old = *address_as_ull, assumed;
            do {
                    assumed = old;
                        old = atomicCAS(address_as_ull, assumed,
                                                    __double_as_longlong(val + __longlong_as_double(assumed)));
                          } while (assumed != old);
              return __longlong_as_double(old);
}
#endif
 
__global__ void sumPayoffKernel(float *d_s, const unsigned N_PATHS, double *mysum)
{
  unsigned idx =  threadIdx.x + blockIdx.x * blockDim.x;
  unsigned stride = blockDim.x * gridDim.x;
  unsigned tid = threadIdx.x;
 
  extern __shared__ double smdata[];
  smdata[tid] = 0.0;
 
  for (unsigned i = idx; i<N_PATHS; i+=stride)
  {
    smdata[tid] += (double) d_s[i];
  }
 
  for (unsigned s=blockDim.x/2; s>0; s>>=1)
  {
    __syncthreads();
    if (tid < s) smdata[tid] += smdata[tid + s];
  }
 
  if (tid == 0)
  {
    atomicAdd(mysum, smdata[0]);
  }
}

__global__ void barrier_option(
    float *d_s,
    const float T,
    const float K,
    const float B,
    const float S0,
    const float sigma,
    const float mu,
    const float r,
    const float * d_normals,
    const long N_STEPS,
    const long N_PATHS)
{
  unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned stride = blockDim.x * gridDim.x;
  const float tmp1 = mu*T/N_STEPS;
  const float tmp2 = exp(-r*T);
  const float tmp3 = sqrt(T/N_STEPS);
  double running_average = 0.0;

  for (unsigned i = idx; i<N_PATHS; i+=stride)
{
  float s_curr = 50;
  for(unsigned n = .0; n < N_STEPS; n++){
      s_curr += tmp1 * s_curr + sigma*s_curr*tmp3*d_normals[i + n * N_PATHS];
      running_average += (s_curr - running_average) / (n + 1.0) ;
      if (running_average <= B){
          break;
      }
  }

  float payoff = (running_average>K ? running_average-K : 0.f);
  d_s[i] = tmp2 * payoff;
 }
}

int main(int argc, char *argv[]) {
  try {
    // declare variables and constants
    size_t N_PATHS = 8192000;
    size_t N_STEPS = 365;
    if (argc >= 2) N_PATHS = atoi(argv[1]);
    if (argc >= 3) N_STEPS = atoi(argv[2]);

    const float T = 1.0f;
    const float K = 110.0f;
    const float B = 100.0f;
    const float S0 = 120.0f;
    const float sigma = 0.35f;
    const float mu = 0.1f;
    const float r = 0.05f;

    double gpu_sum{0.0};

    int devID{0};
    hipDeviceProp_t deviceProps;

    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    print("CUDA device [%s]\n", deviceProps.name);
    print("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProps.name, deviceProps.major, deviceProps.minor);
    // Generate random numbers on the device

    hiprandGenerator_t curandGenerator;
    CHECKCURAND(hiprandCreateGenerator(&curandGenerator, HIPRAND_RNG_PSEUDO_MTGP32));
    CHECKCURAND(hiprandSetPseudoRandomGeneratorSeed(curandGenerator, 1234ULL)) ;

    const size_t N_NORMALS = (size_t)N_STEPS * N_PATHS;
    float *d_normals;
    checkCudaErrors(hipMalloc(&d_normals, N_NORMALS * sizeof(float)));
    CHECKCURAND(hiprandGenerateNormal(curandGenerator, d_normals, N_NORMALS, 0.0f, 1.0f));
    hipDeviceSynchronize();
  
      // before kernel launch, check the max potential blockSize
      int BLOCK_SIZE, GRID_SIZE;
      checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&GRID_SIZE,
                                                         &BLOCK_SIZE,
                                                         barrier_option,
                                                         0, N_PATHS));
      std::cout << "suggested block size " << BLOCK_SIZE
                << " \nsuggested grid size " << GRID_SIZE
                << std::endl;

      std::cout << "Used grid size " << GRID_SIZE << std::endl;

      // Kernel launch
      auto t1=std::chrono::high_resolution_clock::now();

      float *d_s;
      checkCudaErrors(hipMalloc(&d_s, N_PATHS*sizeof(float)));

      auto t3=std::chrono::high_resolution_clock::now();
      barrier_option<<<GRID_SIZE, BLOCK_SIZE>>>(d_s, T, K, B, S0, sigma, mu, r, d_normals, N_STEPS, N_PATHS);
      hipDeviceSynchronize();
      auto t4=std::chrono::high_resolution_clock::now();
 
      double* mySum;
      checkCudaErrors(hipMallocManaged(&mySum, sizeof(double)));
      sumPayoffKernel<<<GRID_SIZE, BLOCK_SIZE, BLOCK_SIZE*sizeof(double)>>>(d_s, N_PATHS, mySum);
      hipDeviceSynchronize();
      auto t5=std::chrono::high_resolution_clock::now();
 
      std::cout << "sumPayoffKernel takes "
                << std::chrono::duration_cast<std::chrono::microseconds>(t5-t4).count() / 1000.f
                << " ms\n";
 
      gpu_sum = mySum[0] / N_PATHS;
 
      auto t2=std::chrono::high_resolution_clock::now();
 
      // clean up
      CHECKCURAND(hiprandDestroyGenerator( curandGenerator )) ;
      checkCudaErrors(hipFree(d_s));
      checkCudaErrors(hipFree(d_normals));
      checkCudaErrors(hipFree(mySum));
 
      std::cout << "price "
              << gpu_sum
              << " time "
                << std::chrono::duration_cast<std::chrono::microseconds>(t5-t1).count() / 1000.f
                << " ms\n";
  }
 
  catch(std::
        exception& e)
  {
    std::cout<< "exception: " << e.what() << "\n";
  }


// Step 1: GPU memory allocated, initialized by CuPY array

output = cupy.zeros(N_PATHS, dtype=cupy.float32)

// Step 2: CuPy random function (cuRAND library), random number generation is defined as (below):

randoms_gpu = cupy.random.normal(0, 1, N_PATHS * N_STEPS, dtype=cupy.float32)

// Step 3: Execute GPU mean value composition (CuPy library) 

v = output.mean()

// Step 4: GPU memory deallocation (automated)



// Execute Monte Carlo simulation on single core CPU, t=41.6s

@njit(fastmath=True)
def cpu_barrier_option(d_s, T, K, B, S0, sigma, mu, r, d_normals, N_STEPS, N_PATHS):
    tmp1 = mu*T/N_STEPS
    tmp2 = math.exp(-r*T)
    tmp3 = math.sqrt(T/N_STEPS)
    running_average = 0.0
    for i in range(N_PATHS):
        s_curr = 50
        for n in range(N_STEPS):
            s_curr += tmp1 * s_curr + sigma*s_curr*tmp3*d_normals[i + n * N_PATHS]
            running_average = running_average + 1.0/(n + 1.0) * (s_curr - running_average)
            if running_average <= B:
                break
        payoff = running_average - K if running_averages>K else 0
        d_s[i] = tmp2 * payoff

// Execute Monte Carlo simulation on single core CPU (DGX-1 Intel CPU)

@njit(fastmath=True, parallel=True)
def cpu_multiplecore_barrier_option(d_s, T, K, B, S0, sigma, mu, r, d_normals, N_STEPS, N_PATHS):
    tmp1 = mu*T/N_STEPS
    tmp2 = math.exp(-r*T)
    tmp3 = math.sqrt(T/N_STEPS)
    for i in prange(N_PATHS):
        s_curr = S0
        running_average = 0.0
        for n in range(N_STEPS):
            s_curr += tmp1 * s_curr + sigma*s_curr*tmp3*d_normals[i + n * N_PATHS]
            running_average = running_average + 1.0/(n + 1.0) * (s_curr - running_average)
            if running_average <= B:
                break
        payoff = running_average - K if running_average>K else 0
        d_s[i] = tmp2 * payoff 

// Numba library approach, single GPU V100 GPU; t=65ms

@cuda.jit
def numba_gpu_barrier_option(d_s, T, K, B, S0, sigma, mu, r, d_normals, N_STEPS, N_PATHS):
    # ii - overall thread index
    ii = cuda.threadIdx.x + cuda.blockIdx.x * cuda.blockDim.x
    stride = cuda.gridDim.x * cuda.blockDim.x
    tmp1 = mu*T/N_STEPS
    tmp2 = math.exp(-r*T)
    tmp3 = math.sqrt(T/N_STEPS)
    running_average = 0.0
    for i in range(ii, N_PATHS, stride):
        s_curr = S0
        for n in range(N_STEPS):
            s_curr += tmp1 * s_curr + sigma*s_curr*tmp3*d_normals[i + n * N_PATHS]
            running_average += (s_curr - running_average) / (n + 1.0)
            if running_average <= B:
                break
        payoff = running_average - K if running_average>K else 0
        d_s[i] = tmp2 * payoff 

// CuPy library approach, single GPU, 29ms

cupy_barrier_option = cupy.RawKernel(r'''
extern "C" __global__ void barrier_option(
    float *d_s,
    const float T,
    const float K,
    const float B,
    const float S0,
    const float sigma,
    const float mu,
    const float r,
    const float * d_normals,
    const long N_STEPS,
    const long N_PATHS)
{
  unsigned idx =  threadIdx.x + blockIdx.x * blockDim.x;
  unsigned stride = blockDim.x * gridDim.x;
  unsigned tid = threadIdx.x;
 
  const float tmp1 = mu*T/N_STEPS;
  const float tmp2 = exp(-r*T);
  const float tmp3 = sqrt(T/N_STEPS);
  double running_average = 0.0;
 
  for (unsigned i = idx; i<N_PATHS; i+=stride)
  {
    float s_curr = S0;
    unsigned n=0;
    for(unsigned n = 0; n < N_STEPS; n++){
       s_curr += tmp1 * s_curr + sigma*s_curr*tmp3*d_normals[i + n * N_PATHS];
       running_average += (s_curr - running_average) / (n + 1.0) ;
       if (running_average <= B){
           break;
       }
    }
 
    float payoff = (running_average>K ? running_average-K : 0.f);
    d_s[i] = tmp2 * payoff;
  }
}
 
''', 'barrier_option')

// Multiple GPUs (Dask)

def get_option_price(T, K, B, S0, sigma, mu, r, N_PATHS = 8192000, N_STEPS = 365, seed=3):
    number_of_threads = 256
    number_of_blocks = (N_PATHS-1) // number_of_threads + 1
    cupy.random.seed(seed)
    randoms_gpu = cupy.random.normal(0, 1, N_PATHS * N_STEPS, dtype=cupy.float32)
    output =  cupy.zeros(N_PATHS, dtype=cupy.float32)
    cupy_barrier_option((number_of_blocks,), (number_of_threads,),
                   (output, np.float32(T), np.float32(K), 
                    np.float32(B), np.float32(S0), 
                    np.float32(sigma), np.float32(mu), 
                    np.float32(r),  randoms_gpu, N_STEPS, N_PATHS))
    v = output.mean()
    out_df = cudf.DataFrame()
    out_df['p'] = cudf.Series([v.item()])
    return out_df 

x = dask_cudf.from_delayed([delayed(get_option_price)(T=1.0, K=110.0,  B=100.0, S0=120.0, sigma=0.35, mu=0.1, r=0.05, seed=3000+i) for i in  range(1600)])
x.mean().compute()
x.std().compute() 







